#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

constexpr unsigned int THREADS_PER_BLOCK_DIM = 8; // 8x8 threads in a block

hipError_t error = hipSuccess;

__global__ void fillData(const unsigned int pitch, const unsigned int rows, const unsigned int cols, float* data)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid out-of-bounds memory access
    if (x < cols && y < rows)
    {
        // Calculate the offset to the correct element based on pitch
        float* row = (float*)((char*)data + y * pitch);
        row[x] = y * cols + x; // Fill data with incremental values
    }
}

int main(int argc, char* argv[])
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    hipSetDevice(0);

    float* devPtr;
    float* hostPtr;
    size_t pitch;

    const unsigned int mRows = 5;
    const unsigned int mCols = 10;

    // Allocate pitch memory with alignment using hipMallocPitch
    hipError_t err = hipMallocPitch(&devPtr, &pitch, mCols * sizeof(float), mRows);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA malloc failed!" << std::endl;
        return -1;
    }

    // Prepare grid and blocks (2D grid of 2D blocks of size 8x8)
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);
    dim3 numBlocks((mCols + THREADS_PER_BLOCK_DIM - 1) / THREADS_PER_BLOCK_DIM,
        (mRows + THREADS_PER_BLOCK_DIM - 1) / THREADS_PER_BLOCK_DIM);

    // Launch the kernel
    fillData << <numBlocks, threadsPerBlock >> > (pitch, mRows, mCols, devPtr);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Allocate host memory for the matrix
    hostPtr = (float*)malloc(mRows * mCols * sizeof(float));

    // Copy data back from device to host using hipMemcpy2D
    hipMemcpy2D(hostPtr, mCols * sizeof(float), devPtr, pitch, mCols * sizeof(float), mRows, hipMemcpyDeviceToHost);

    // Check the data by printing out the matrix
    std::cout << "Matrix (after kernel incrementing):" << std::endl;
    for (unsigned int i = 0; i < mRows; ++i)
    {
        for (unsigned int j = 0; j < mCols; ++j)
        {
            std::cout << hostPtr[i * mCols + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device and host memory
    hipFree(devPtr);
    free(hostPtr);

    return 0;
}
