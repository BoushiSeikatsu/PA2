#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <random>

//WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
constexpr unsigned int TPB = 128;
constexpr unsigned int NO_FORCES = 256;
constexpr unsigned int NO_RAIN_DROPS = 1 << 20;

constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using namespace std;

__host__ float3 *createData(const unsigned int length)
{
	//TODO: Generate float3 vectors. You can use 'make_float3' method.
	random_device rd;
	mt19937_64 mt(rd());
	uniform_real_distribution<float> dist(0.0f, 1.0f);
	float3* data = static_cast<float*>(::new_handler(length * sizeof(float3)));
	float3* ptr;
	return data;
}

__host__ void printData(const float3 *data, const unsigned int length)
{
	if (data == 0) return;
	const float3 *ptr = data;
	for (unsigned int i = 0; i<length; i++, ptr++)
	{
		printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Sums the forces to get the final one using parallel reduction. 
/// 		    WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
/// <param name="dForces">	  	The forces. </param>
/// <param name="noForces">   	The number of forces. </param>
/// <param name="dFinalForce">	[in,out] If non-null, the final force. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void reduce(const float3 * __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce)
{
	__shared__ float3 sForces[TPB];					//SEE THE WARNING MESSAGE !!!
	unsigned int tid = threadIdx.x;
	unsigned int next = TPB;						//SEE THE WARNING MESSAGE !!!

	float3* src = &sForces[tid];
	*src = dForces[tid];
	float3* src2 = (float3*)&dForces[tid + next];
	src->x += src2->x;
	src->y += src2->y;
	src->z += src2->z;
	__syncthreads();
	next >>= 1;
	if (tid >= next) return;
	src2 = src + next;
	src->x += src2->x;
	src->y += src2->y;
	src->z += src2->z;
	__syncthreads();
	next >>= 1;
	if (tid >= next) return;
	volatile float3* vscr = &sForces[tid];
	volatile float3* vscr2 = vscr + next;
	vscr->x += vscr2->x;
	vscr->y += vscr2->y;
	vscr->z += vscr2->z;

	//TODO: Make the reduction
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds the FinalForce to every Rain drops position. </summary>
/// <param name="dFinalForce">	The final force. </param>
/// <param name="noRainDrops">	The number of rain drops. </param>
/// <param name="dRainDrops"> 	[in,out] If non-null, the rain drops positions. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops)
{
	//TODO: Add the FinalForce to every Rain drops position.
}


int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	float3 *hForces = createData(NO_FORCES);
	float3 *hDrops = createData(NO_RAIN_DROPS);

	float3 *dForces = nullptr;
	float3 *dDrops = nullptr;
	float3 *dFinalForce = nullptr;

	checkCudaErrors(hipMalloc((void**)&dForces, NO_FORCES * sizeof(float3)));
	checkCudaErrors(hipMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3)));
	checkCudaErrors(hipMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&dFinalForce, sizeof(float3)));

	KernelSetting ksReduce;

	//TODO: ... Set ksReduce
	
	KernelSetting ksAdd;
	//TODO: ... Set ksAdd
	
	for (unsigned int i = 0; i<1000; i++)
	{
		reduce<<<ksReduce.dimGrid, ksReduce.dimBlock>>>(dForces, NO_FORCES, dFinalForce);
		add<<<ksAdd.dimGrid, ksAdd.dimBlock>>>(dFinalForce, NO_RAIN_DROPS, dDrops);
	}

	checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
	// checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");

	if (hForces)
		free(hForces);
	if (hDrops)
		free(hDrops);

	checkCudaErrors(hipFree(dForces));
	checkCudaErrors(hipFree(dDrops));

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	printf("Time to get device properties: %f ms", elapsedTime);
}
